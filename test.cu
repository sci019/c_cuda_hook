#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>

// カーネル関数の定義 (__global__ 修飾子を付ける)
__global__ void addKernel(int *data) {
    int idx = threadIdx.x;  // スレッドIDを取得
    data[idx] = idx;         // 各スレッドが1を加算
}

int main() {
    const int arraySize = 1024;
    int hostData[arraySize];
    for (int i ; i < arraySize ; i++){
        hostData[i] = 0;
    }
    int *deviceData;

    // GPU上にメモリを確保 (hipMalloc)
    hipMalloc((void**)&deviceData, arraySize * sizeof(int));
    hipMemcpy(deviceData, hostData, arraySize * sizeof(int), hipMemcpyHostToDevice);

    // カーネルを起動 (<<<1, arraySize>>> はスレッド数を指定)
    addKernel<<<1, arraySize>>>(deviceData);
    hipDeviceSynchronize();

    // 結果をCPUにコピー (hipMemcpy)
    hipMemcpy(hostData, deviceData, arraySize * sizeof(int), hipMemcpyDeviceToHost);

    // 結果を表示
    int err = 0;
    for (int i = 0; i < arraySize; i++) {
        if (i != hostData[i]) {
            err +=1;
        }
    }
    std::cout << "err: " << err << std::endl;
    // std::cout << "Result[" << arraySize - 1 << "]: " << hostData[arraySize - 1] << std::endl;

    // GPUメモリを解放
    hipFree(deviceData);

    return 0;
}